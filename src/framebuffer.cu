#include "common.cuh"
#include "framebuffer.cuh"

namespace rt {

hipSurfaceObject_t Framebuffer::_GetMappedSurface() {
    hipArray_t hipArray;
    CUDA_CHECK(hipGraphicsMapResources(1, &_cuda_texture_resource));
    CUDA_CHECK(hipGraphicsSubResourceGetMappedArray(&hipArray, _cuda_texture_resource, 0, 0));

    _cuda_res_desc.resType = hipResourceTypeArray;
    _cuda_res_desc.res.array.array = hipArray;
    _cuda_res_desc.flags = 0;

    CUDA_CHECK(hipCreateSurfaceObject(&_surface, &_cuda_res_desc));
    return _surface;
}

void Framebuffer::_UnmapSurface() {
    CUDA_CHECK(hipDestroySurfaceObject(_surface));
    CUDA_CHECK(hipGraphicsUnmapResources(1, &_cuda_texture_resource));
}

}  // namespace rt