#include "hip/hip_runtime.h"
#include <cuda_gl_interop.h>
#include "common.hu"
#include "framebuffer.hu"
#include "renderer.hu"

namespace rt {

DEVICE Vec3f TraceRay(const Rayf& ray, const Sphere* spheres, int num_spheres, int max_bounces,
                      hiprandState* random_state);

KERNEL void InitRandomStatesKernel(hiprandState* random_states, int num_states,
                                   unsigned long long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < num_states) {
        hiprand_init(seed, idx, 0, &random_states[idx]);
    }
}

KERNEL void RenderKernel(hipSurfaceObject_t surface, int width, int height, Camera camera,
                         const Sphere* spheres, int num_spheres, int max_bounces,
                         hiprandState* random_states, Vec3f* accumulated_colors, int sample_count) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    hiprandState* local_state = &random_states[y * width + x];

    // Convert pixel coordinates to UV coordinates
    float u = float(x) / float(width - 1);
    float v = float(height - 1 - y) / float(height - 1);

    Rayf ray = camera.GetRay(u, v);
    Vec3f color = TraceRay(ray, spheres, num_spheres, max_bounces, local_state);

    int pixel_index = y * width + x;
    accumulated_colors[pixel_index] = accumulated_colors[pixel_index] + color;
    Vec3f average_color = accumulated_colors[pixel_index] / float(sample_count);

    surf2Dwrite(make_float4(average_color.x, average_color.y, average_color.z, 1.0f), surface,
                x * sizeof(float4), y);
}

DEVICE Vec3f TraceRay(const Rayf& initial_ray, const Sphere* spheres, int num_spheres,
                      int max_bounces, hiprandState* random_state) {
    Rayf ray = initial_ray;
    Vec3f final_attenuation(1.0f, 1.0f, 1.0f);

    for (int bounce = 0; bounce < max_bounces; ++bounce) {
        bool found_hit = false;
        Hit hit;
        for (int i = 0; i < num_spheres; i++) {
            if ((found_hit = spheres[i].Hit(ray, 0.001f, INFINITY, hit))) {
                break;
            }
        }

        if (found_hit) {
            Rayf scattered;
            Vec3f attenuation;
            if (hit.material->Scatter(ray, hit, attenuation, scattered, random_state)) {
                ray = scattered;
                final_attenuation *= attenuation;
            } else {
                return final_attenuation;
            }
        } else {
            float t = 0.5f * (ray.direction.y + 1.0f);
            Vec3f sky_color = Vec3f(1.0f, 1.0f, 1.0f) * (1.0f - t) + Vec3f(0.5f, 0.7f, 1.0f) * t;
            return final_attenuation * sky_color;
        }
    }

    return Vec3f(0, 0, 0);
}

Renderer::Renderer(int w, int h, int max_bounces)
    : width(w), height(h), max_bounces(max_bounces), sample_count(0),
      _device_random_states(nullptr), _device_accumulator(nullptr) {}

Renderer::~Renderer() {
    Cleanup();
}

bool Renderer::Init() {
    CUDA_CHECK(hipMalloc(&_device_random_states, width * height * sizeof(hiprandState)));
    CUDA_CHECK(hipMalloc(&_device_accumulator, width * height * sizeof(Vec3f)));

    InitRandomStates();

    sample_count = 0;
    return true;
}

void Renderer::Cleanup() {
    if (_device_random_states) {
        hipFree(_device_random_states);
        _device_random_states = nullptr;
    }

    if (_device_accumulator) {
        hipFree(_device_accumulator);
        _device_accumulator = nullptr;
    }
}

void Renderer::InitRandomStates() {
    InitRandomStatesKernel<<<(width * height + 255) / 256, 256>>>(_device_random_states,
                                                                  width * height, time(nullptr));
}

void Renderer::ClearAccumulator() {
    CUDA_CHECK(hipMemset(_device_accumulator, 0, width * height * sizeof(Vec3f)));
    sample_count = 0;
}

void Renderer::RenderFrame(Framebuffer& framebuffer, const Camera& camera, const Sphere* spheres,
                           int num_spheres) {
    sample_count++;
    framebuffer.Map();

    // Launch ray tracing kernel
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x,
                  (height + blockSize.y - 1) / blockSize.y);

    RenderKernel<<<gridSize, blockSize>>>(framebuffer.GetSurface(), width, height, camera, spheres,
                                          num_spheres, max_bounces, _device_random_states,
                                          _device_accumulator, sample_count);

    framebuffer.Unmap();
    CUDA_CHECK(hipStreamSynchronize(0));
}

}  // namespace rt
